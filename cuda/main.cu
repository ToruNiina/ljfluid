#include "hip/hip_runtime.h"
#include <cuda/particle.cuh>
#include <cuda/boundary_condition.cuh>
#include <cuda/grid.cuh>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/inner_product.h>
#include <thrust/functional.h>
#include <thrust/for_each.h>
#include <hiprand.h>
#include <iterator>
#include <iostream>
#include <fstream>
#include <cassert>
#include <cmath>

namespace lj
{
__device__ __host__ constexpr float sgm() noexcept {return 1.0f;}
__device__ __host__ constexpr float eps() noexcept {return 1.0f;}

struct kinetic_energy_calculator
    : thrust::binary_function<float, float4, float>
{
    __device__ __host__
    float operator()(float m, float4 v) const noexcept
    {
        return m * length_sq(v);
    };
};

float kinetic_energy(const particle_container& ps)
{
    return thrust::inner_product(
            ps.device_masses.cbegin(),      ps.device_masses.cend(),
            ps.device_velocities.cbegin(), 0.0, thrust::plus<float>(),
            kinetic_energy_calculator()) * 0.5;
}

struct velocity_verlet_update_1
{
    velocity_verlet_update_1(float dt_, periodic_boundary b_)
        : dt(dt_), dt_half(dt_ * 0.5), b(b_)
    {}

    template<typename Tuple>
    __device__ __host__
    void operator()(Tuple mpvf) const noexcept
    {
        thrust::get<1>(mpvf) = adjust_position(
                thrust::get<1>(mpvf) + dt * thrust::get<2>(mpvf) +
                (dt * dt_half / thrust::get<0>(mpvf)) * thrust::get<3>(mpvf),
                b);

        thrust::get<2>(mpvf) = thrust::get<2>(mpvf) +
                (dt_half / thrust::get<0>(mpvf)) * thrust::get<3>(mpvf);
        return;
    }

    const float dt;
    const float dt_half;
    const periodic_boundary b;
};

struct velocity_verlet_update_2
{
    velocity_verlet_update_2(float dt_)
        : dt(dt_), dt_half(dt_ * 0.5)
    {}

    template<typename Tuple>
    __device__ __host__
    void operator()(Tuple mvf) const noexcept
    {
        thrust::get<1>(mvf) = thrust::get<1>(mvf) +
                (dt_half / thrust::get<0>(mvf)) * thrust::get<2>(mvf);
        thrust::get<2>(mvf) = make_float4(0,0,0,0);
        return;
    }

    const float dt;
    const float dt_half;
};

} // lj

struct tuple_vector_converter
{
    __device__ __host__
    float4 operator()(const thrust::tuple<float, float, float>& t) const noexcept
    {
        return make_float4(
                thrust::get<0>(t), thrust::get<1>(t), thrust::get<2>(t), 0.0);
    };
};

struct position_initializer
{
    __device__ __host__
    float4 operator()(std::size_t i) const noexcept
    {
        return make_float4(1.0 + 2.0 * ((i & 0b000011) >> 0),
                           1.0 + 2.0 * ((i & 0b001100) >> 2),
                           1.0 + 2.0 * ((i & 0b110000) >> 4),
                           0.0);
    };
};

int main()
{
    const float4 upper    = make_float4( 8.0,  8.0,  8.0, 0.0);
    const float4 lower    = make_float4( 0.0,  0.0,  0.0, 0.0);
    const auto   boundary = lj::make_boundary(lower, upper);

    const std::size_t step = 100000;
    const std::size_t N    = std::pow(4, 3);
    const std::size_t seed = 123456789;
    const float kB  = 1.986231313e-3;
    const float T   = 300.0;
    const float dt  = 0.01;

    lj::particle_container ps(N);

    /* initialization */{
        thrust::fill(ps.device_masses.begin(), ps.device_masses.end(), 1.0f);
        thrust::fill(ps.host_masses.begin(),   ps.host_masses.end(),   1.0f);

        thrust::transform(
            /* input  begin */ thrust::make_counting_iterator<std::size_t>(0),
            /* input  end   */ thrust::make_counting_iterator<std::size_t>(N),
            /* output begin */ ps.device_positions.begin(),
            /* conversion   */ position_initializer());

        // prepair cuRAND generators
        hiprandGenerator_t rng;
        const auto st_gen = hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT);
        assert(st_gen  == HIPRAND_STATUS_SUCCESS);
        const auto st_seed = hiprandSetPseudoRandomGeneratorSeed(rng, seed);
        assert(st_seed == HIPRAND_STATUS_SUCCESS);

        thrust::device_vector<float> boltz_x(N);
        thrust::device_vector<float> boltz_y(N);
        thrust::device_vector<float> boltz_z(N);
        {
            const auto st_genrnd = hiprandGenerateNormal(
                    rng, boltz_x.data().get(), N, 0.0, std::sqrt(kB * T));
            assert(st_genrnd == HIPRAND_STATUS_SUCCESS);
        }

        {
            const auto st_genrnd = hiprandGenerateNormal(
                    rng, boltz_y.data().get(), N, 0.0, std::sqrt(kB * T));
            assert(st_genrnd == HIPRAND_STATUS_SUCCESS);
        }

        {
            const auto st_genrnd = hiprandGenerateNormal(
                    rng, boltz_z.data().get(), N, 0.0, std::sqrt(kB * T));
            assert(st_genrnd == HIPRAND_STATUS_SUCCESS);
        }

        thrust::transform(
            thrust::make_zip_iterator(thrust::make_tuple(
                boltz_x.begin(), boltz_y.begin(), boltz_z.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(
                boltz_x.end(),   boltz_y.end(),   boltz_z.end())),
            ps.device_velocities.begin(), tuple_vector_converter());
    }

    ps.pull_device_particles();

    {
        std::ofstream traj("traj.xyz");
        traj << ps.host_positions.size() << "\n\n";
        for(auto iter = ps.host_positions.begin(), iend = ps.host_positions.end();
                iter != iend; ++iter)
        {
            const auto& v = *iter;
            traj << "H      " << std::fixed << std::setprecision(5) << std::showpoint
                 << std::setw(10) << std::right << v.x
                 << std::setw(10) << std::right << v.y
                 << std::setw(10) << std::right << v.z << '\n';
        }
    }

    std::cerr << "kinetic energy = " << lj::kinetic_energy(ps)
              << ", 3/2 NkBT = " << N * kB * T * 1.5 << std::endl;

    //TODO: add potential
    lj::grid grid(lj::sgm() * 3, boundary);
    std::cerr << grid.Nx << std::endl;
    std::cerr << grid.Ny << std::endl;
    std::cerr << grid.Nz << std::endl;

    grid.assign(ps.device_positions);

    std::cerr << "assigned" << std::endl;

    for(std::size_t i=0; i<grid.cell.size()-1; ++i)
    {
        const auto rg = grid.get_range(i);
        std::cerr << '{' << rg.first << ", " << rg.second << "}, ";
    }
    std::cerr << std::endl;

//
//     std::size_t idx = 0;
//     thrust::host_vector<lj::array<std::size_t, 27>> adjs = grid.adjs;
//     for(auto iter = adjs.begin(), iend = adjs.end(); iter != iend; ++iter)
//     {
//         std::cerr << idx << '\n';
//         for(std::size_t i=0; i<27; ++i)
//         {
//             std::cerr << (*iter)[i] << ',';
//         }
//         std::cerr << "\n\n";
//         ++idx;
//     }
/*
    const lj::velocity_verlet_update_1 update1(dt, boundary);
    const lj::velocity_verlet_update_2 update2(dt);
    for(std::size_t s=0; s < step; ++s)
    {
        if(s % 1000 == 0)
        {
            ps.pull_device_particles();

            std::ofstream traj("traj.xyz",
                    std::ios_base::app | std::ios_base::out);

            traj << ps.host_positions.size() << "\n\n";
            for(auto iter = ps.host_positions.begin(), iend = ps.host_positions.end();
                    iter != iend; ++iter)
            {
                const auto& v = *iter;
                traj << "H      " << std::fixed << std::setprecision(5) << std::showpoint
                     << std::setw(10) << std::right << v.x
                     << std::setw(10) << std::right << v.y
                     << std::setw(10) << std::right << v.z << '\n';
            }
        }

        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(
                ps.device_masses.begin(), ps.device_positions.begin(),
                ps.device_velocities.begin(), ps.device_forces.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(
                ps.device_masses.end(), ps.device_positions.end(),
                ps.device_velocities.end(), ps.device_forces.end())),
            update1);

        // calc force here

        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(
                ps.device_masses.begin(),
                ps.device_velocities.begin(), ps.device_forces.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(
                ps.device_masses.end(),
                ps.device_velocities.end(), ps.device_forces.end())),
            update2);
    } */

    return 0;
}
